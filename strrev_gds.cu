#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <fcntl.h>
#include <unistd.h>
#include <hip/hip_runtime.h>


#include "cufile.h"

#define INIT_BUFSIZE 4096
#define TESTFILE "/mnt/test"

__global__ void hello(char *str) {
	printf("Hello World!\n");
	printf("buf: %s\n", str);
}

__global__ void strrev(char *str) {
	int size = 0;
	while (str[size] != '\0') {
		size++;
	}
	for(int i=0;i<size/2;++i) {
		char t = str[i];
		str[i] = str[size-1-i];
		str[size-1-i] = t;
	}
	/*
	printf("buf: %s\n", str);
	printf("size: %d\n", size);
	*/
}

int main(int argc, char *argv[])
{
	int fd;
	int ret;
	char *system_buf;
	char *gpumem_buf;
	long buf_size = INIT_BUFSIZE;
	system_buf = (char*)malloc(buf_size);
	hipMalloc(&gpumem_buf, buf_size);
        off_t file_offset = 0;
        off_t mem_offset = 0;
	CUfileDescr_t cf_desc; 
	CUfileHandle_t cf_handle;

	cuFileDriverOpen();
	fd = open(argv[1], O_RDWR | O_DIRECT, 0664);

	cf_desc.handle.fd = fd;
	cf_desc.type = CU_FILE_HANDLE_TYPE_OPAQUE_FD;

	cuFileHandleRegister(&cf_handle, &cf_desc);
	cuFileBufRegister((char*)gpumem_buf, buf_size, 0);

	ret = cuFileRead(cf_handle, (char*)gpumem_buf, buf_size, file_offset, mem_offset);
	if (ret < 0) {
		printf("cuFileRead failed : %d", ret); 
	}

	/*
	hello<<<1,1>>>(gpumem_buf);
	*/
	strrev<<<1,1>>>(gpumem_buf);

	/*
	ret = cuFileWrite(cf_handle, (char*)gpumem_buf, buf_size, file_offset, mem_offset);
	if (ret < 0) {
		printf("cuFileWrite failed : %d", ret); 
	}
	*/

	hipMemcpy(system_buf, gpumem_buf, buf_size, hipMemcpyDeviceToHost);
	printf("%s: %s\n", TESTFILE, system_buf);

	cuFileBufDeregister((char*)gpumem_buf);

	hipFree(gpumem_buf);
	free(system_buf);

	close(fd);
	cuFileDriverClose();
}
