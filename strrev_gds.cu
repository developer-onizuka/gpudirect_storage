#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <fcntl.h>
#include <unistd.h>
#include <hip/hip_runtime.h>


#include "cufile.h"

#define KB(x) ((x)*1024L)
#define TESTFILE "/mnt/test"

__global__ void hello(char *str) {
	printf("Hello World!\n");
	printf("buf: %s\n", str);
}

__global__ void strrev(char *str, int *len) {
	int size = 0;
	while (str[size] != '\0') {
		size++;
	}
	for(int i=0;i<size/2;++i) {
		char t = str[i];
		str[i] = str[size-1-i];
		str[size-1-i] = t;
	}
	/*
	printf("buf: %s\n", str);
	printf("size: %d\n", size);
	*/
	*len = size;
}

int main(int argc, char *argv[])
{
	int fd;
	int ret;
	int *sys_len;
	int *gpu_len;
	char *system_buf;
	char *gpumem_buf;
	system_buf = (char*)malloc(KB(4));
	sys_len = (int*)malloc(KB(1));
	hipMalloc(&gpumem_buf, KB(4));
	hipMalloc(&gpu_len, KB(1));
        off_t file_offset = 0;
        off_t mem_offset = 0;
	CUfileDescr_t cf_desc; 
	CUfileHandle_t cf_handle;

	cuFileDriverOpen();
	fd = open(argv[1], O_RDWR | O_DIRECT);

	cf_desc.handle.fd = fd;
	cf_desc.type = CU_FILE_HANDLE_TYPE_OPAQUE_FD;

	cuFileHandleRegister(&cf_handle, &cf_desc);
	cuFileBufRegister((char*)gpumem_buf, KB(4), 0);

	ret = cuFileRead(cf_handle, (char*)gpumem_buf, KB(4), file_offset, mem_offset);
	if (ret < 0) {
		printf("cuFileRead failed : %d", ret); 
	}

	/*
	hello<<<1,1>>>(gpumem_buf);
	*/
	strrev<<<1,1>>>(gpumem_buf, gpu_len);

	hipMemcpy(sys_len, gpu_len, KB(1), hipMemcpyDeviceToHost);
	printf("sys_len : %d\n", *sys_len); 
	ret = cuFileWrite(cf_handle, (char*)gpumem_buf, *sys_len, file_offset, mem_offset);
	if (ret < 0) {
		printf("cuFileWrite failed : %d", ret); 
	}

	hipMemcpy(system_buf, gpumem_buf, KB(4), hipMemcpyDeviceToHost);
	printf("%s\n", system_buf);
	printf("See also %s\n", argv[1]);

	cuFileBufDeregister((char*)gpumem_buf);

	hipFree(gpumem_buf);
	hipFree(gpu_len);
	free(system_buf);
	free(sys_len);

	close(fd);
	cuFileDriverClose();
}
